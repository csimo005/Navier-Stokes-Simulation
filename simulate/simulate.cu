#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DELTA_T 0.01667
#define LATTICE_SIZE 0.01

#include "advection/advection.h"
#include "diffusion/diffusion.h"
#include "force/force.h"
#include "projection/project.h"
#include "points/points.h"

#include "util.h"

int main(int argc, char *argv[]) {
    unsigned int size_x=200;
    unsigned int size_y=200;
    unsigned int size_z=200;

    unsigned int n=3*size_x*size_y*size_z;
    unsigned int n_p = 7000000; //1 million points defined by 7 flos each

    float *u, *p;
//    hiprandState_t *states; 

    hipMalloc((void **) &u, n*sizeof(float));
    hipMallocManaged(&p, n_p*sizeof(float));

    float *force_h, *grav_h, *pos_h;
    float *force_d, *grav_d, *pos_d;

    force_h = (float*) malloc(3*sizeof(float));
    pos_h   = (float*) malloc(3*sizeof(float));
    grav_h  = (float*) malloc(3*sizeof(float));

    force_h[0] = 0.5; grav_h[0] =  0;   pos_h[0] = 1.25;
    force_h[1] = 0.5; grav_h[1] = -9.8; pos_h[1] = 1.25;
    force_h[2] = 0.5; grav_h[2] =  0;   pos_h[2] = 1.25;

    hipMalloc((void**) &force_d, 3*sizeof(float));
    hipMalloc((void**) &grav_d, 3*sizeof(float));
    hipMalloc((void**) &pos_d, 3*sizeof(float));
    hipError_t cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) {
        printf("Error: failed to allocated device variables\n\tThrew: %s\n",hipGetErrorString(cuda_ret));fflush(stdout);
	return 0;
    }

    cuda_ret = hipMemcpy(force_d, force_h, 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(grav_d, grav_h, 3*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(pos_d, pos_h, 3*sizeof(float), hipMemcpyHostToDevice);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) {
        printf("Error: Memcpy of initial values failed\n\tThrew: %s\n",hipGetErrorString(cuda_ret));fflush(stdout);
	return 0;
    }


    dim3 blockDim_1D(1000,1,1);
    dim3 gridDim_1D((n-1)/1000+1,1,1);

    zeroVector<<<gridDim_1D, blockDim_1D>>>(u,n);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) {
        printf("Error: Failed to initialize velocity field\n\tThrew: %s\n",hipGetErrorString(cuda_ret));fflush(stdout);
        return 0;
    }

    initPoints(p, 1000000, size_x, size_y, size_z); 

    for(unsigned int i=0;i<11/DELTA_T;i++) {
        globalForce(u,grav_d,size_x,size_y,size_z);
	hipDeviceSynchronize();
	if(i > 1/DELTA_T && i < 3/DELTA_T) {
            localForce(u,force_d,pos_d,0.25,size_x,size_y,size_z);
	}
	hipDeviceSynchronize();
	
        advection(u,size_x,size_y,size_z);
        diffusion(u,size_x,size_y,size_z);
	project(u,size_x,size_y,size_z);
        hipDeviceSynchronize();
        updatePoints(u, p, 1000000, size_x, size_y, size_z);

	printf("%.2f\n", i*DELTA_T); fflush(stdout);
    }

    hipFree(u);
    hipFree(force_d);
    hipFree(pos_d);
    hipFree(grav_d);
    hipFree(p);

    free(force_h);
    free(pos_h);
    free(grav_h);

    return 0;
}
